#include "hip/hip_runtime.h"
/*
Copyright (c) 2013, Los Alamos National Security, LLC
All rights reserved.
Copyright 2013. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <fstream>
#include <iostream>
#include <sstream>
#include <float.h>

#include "CoGLRender.h"

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)


//------------------------------------------------------------------------------
// CoGLRender::CoGLRender()
//
// Constructor for CoGLRender class
//------------------------------------------------------------------------------ 
CoGLRender::CoGLRender()
{
}


//---------------------------------------------------------------------------
// CoGLRender::~CoGLRender()
//
// Destructor of CoGLRender class
//---------------------------------------------------------------------------
CoGLRender::~CoGLRender()
{
    #ifdef USE_INTEROP
      if (vbo_buffers[0])
      {
	for (int i=0; i<3; i++) hipGraphicsUnregisterResource(vbo_resources[i]);
	for (int i=0; i<3; i++)
	{
	  glBindBuffer(1, vbo_buffers[i]);
	  glDeleteBuffers(1, &(vbo_buffers[i]));
	  vbo_buffers[i] = 0;
	}
      }     
    #else
      vertices.clear(); normals.clear(); colors.clear(); 
    #endif
}


//---------------------------------------------------------------------------
// CoGLRender::initialize()
//
// Initialize OpenGL settings, and call initialize_simulation
//---------------------------------------------------------------------------
void CoGLRender::initialize()
{
    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    float white[] = { 0.5, 0.5, 0.5, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float light_pos[] = { 0.0, 0.0, 10.0, 1.0 };
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, light_pos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);

    #ifdef USE_INTEROP
      glewInit();
      cudaGLSetGLDevice(0);

      // initialize contour buffer objects
      glGenBuffers(3, vbo_buffers);
      for (int i=0; i<3; i++)
      {
        unsigned int buffer_size = DIM3*VERTICES_PER_CELL*sizeof(float4);
        glBindBuffer(GL_ARRAY_BUFFER, vbo_buffers[i]);
        glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
      }

      glBindBuffer(GL_ARRAY_BUFFER, 0);
      for (int i=0; i<3; i++) hipGraphicsGLRegisterBuffer(&(vbo_resources[i]), vbo_buffers[i], cudaGraphicsMapFlagsWriteDiscard);
    #endif

    initialize_simulation();

    rotate(0.2, -0.2);
    max_camera_fov = 40.0;  zoom_pct = 0.5;
}


//---------------------------------------------------------------------------
// CoGLRender::display()
//
// Render the structured grid, with colors based on the deviatoric strain
//---------------------------------------------------------------------------
void CoGLRender::display()
{
    time_step();

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(max_camera_fov*zoom_pct, 1.0, 1.0, 1000.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0, 0, 8.0f*DIM, 0, 0, 0, 0, 1, 0);
    glPushMatrix();

    glTranslatef(-DIM/2.0f, -DIM/2.0f, -DIM/2.0f);
    qrot.getRotMat(rotation_matrix);
    glMultMatrixf(rotation_matrix);

    float center_x = DIM/2.0f;  float center_y = DIM/2.0f;  float center_z = DIM/2.0f;
    GLfloat matrix[16];
    glGetFloatv(GL_MODELVIEW_MATRIX, matrix);
    float offset_x = matrix[0]*center_x + matrix[1]*center_y + matrix[2]*center_z;
    float offset_y = matrix[4]*center_x + matrix[5]*center_y + matrix[6]*center_z;
    float offset_z = matrix[8]*center_x + matrix[9]*center_y + matrix[10]*center_z;
    offset_x = center_x - offset_x; offset_y = center_y - offset_y; offset_z = center_z - offset_z;
    glTranslatef(-offset_x, -offset_y, -offset_z);

    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
    glEnableClientState(GL_NORMAL_ARRAY);
  
    #ifdef USE_INTEROP
      glBindBuffer(GL_ARRAY_BUFFER, vbo_buffers[0]);     
      glVertexPointer(3, GL_FLOAT, 0, 0);     
      glBindBuffer(GL_ARRAY_BUFFER, vbo_buffers[1]);
      glColorPointer(4, GL_FLOAT, 0, 0);
      glBindBuffer(GL_ARRAY_BUFFER, vbo_buffers[2]);
      glNormalPointer(GL_FLOAT, 0, 0); 
      glDrawArrays(GL_QUADS, 0, simulation->n_vertices);   
      glBindBuffer(GL_ARRAY_BUFFER, 0);
    #else       
      if (cur_timestep < max_timesteps) colors.assign(simulation->colors_begin(), simulation->colors_end());
          
      glNormalPointer(GL_FLOAT, 0, &normals[0]);
      glColorPointer(4, GL_FLOAT, 0, &colors[0]);
      glVertexPointer(3, GL_FLOAT, 0, &vertices[0]);
      glDrawArrays(GL_QUADS, 0, simulation->n_vertices);   
    #endif

    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    glDisableClientState(GL_NORMAL_ARRAY);

    glPopMatrix();
}


//------------------------------------------------------------------------------
// CoGLRender::rotate()
//
// param    float a_angle1   Rotation about first axis
// param    float a_angle2   Rotation about second axis
//
// Rotate view based on input angles
//------------------------------------------------------------------------------
void CoGLRender::rotate(float a_angle1, float a_angle2)
{
    Quaternion new_rot_x;
    new_rot_x.setEulerAngles(a_angle1, 0.0, 0.0);
    qrot.mul(new_rot_x);

    Quaternion new_rot_y;
    new_rot_y.setEulerAngles(0.0, 0.0, a_angle2);
    qrot.mul(new_rot_y);
}


//------------------------------------------------------------------------------
// CoGLRender::zoom()
//
// param    float a_delta    Change to percentage of max FOV to use for field of view angle
//
// Increment the zoom level based on input parameter
//------------------------------------------------------------------------------
void CoGLRender::zoom(float a_delta)
{
    zoom_pct += a_delta;
    if (zoom_pct > 1.0) zoom_pct = 1.0;  if (zoom_pct < 0.0) zoom_pct = 0.0;
}


//---------------------------------------------------------------------------
// CoGLRender::initialize_simulation()
//
// Initialize simulation parameters, read input data, and pass to CoGLSim class
//---------------------------------------------------------------------------
void CoGLRender::initialize_simulation()
{
    simulation = new CoGLSim<double>(DIM);

    #ifdef USE_INTEROP
      for (int i=0; i<3; i++) simulation->vbo_resources[i] = vbo_resources[i];
    #endif

    simulation->create_mesh();

    #ifndef USE_INTEROP
      normals.assign(simulation->normals_begin(), simulation->normals_end());
      vertices.assign(simulation->vertices_begin(), simulation->vertices_end());
      colors.assign(simulation->colors_begin(), simulation->colors_end());
    #endif 
  
    h_ux.resize(DIM3);    h_uy.resize(DIM3);    h_uz.resize(DIM3);
    h_uxdt.resize(DIM3);  h_uydt.resize(DIM3);  h_uzdt.resize(DIM3);
    h_uxx_applied.resize(DIM3);
    
    max_timesteps=200000;  temp=255.0;  d2=1.0;
    eta=10.0;  delta=0.01;  h=1.0;  
    shear_a=28.0*pow(10.0,10);  bulk_a=14.0*pow(10.0,10);
    ao=1.97*pow(10.0,10);
    
    as = shear_a/ao;  ac = bulk_a/ao;
    printf("AS=%lf, AC=%lf\n", as, ac);

    tau=(temp-270.)/(295.-270.);
    eta0_xx=(3.795-3.756)/3.756;
    eta0_yy=eta0_xx;
    eta0_zz=(3.725-3.756)/3.756;
    e1_0=(eta0_xx+eta0_yy+eta0_zz)/sqrt(3.);
    e3_0=(eta0_xx+eta0_yy-2.0*eta0_zz)/sqrt(6.);
    ao=1.97*pow(10.0,10);
    eo=-0.5000*ac*e1_0/e3_0;
   
    char fuxFilename[1024], fuyFilename[1024], fuzFilename[1024];
    sprintf(fuxFilename, "%s/fort.31", STRINGIZE_VALUE_OF(DATA_DIRECTORY));
    sprintf(fuyFilename, "%s/fort.32", STRINGIZE_VALUE_OF(DATA_DIRECTORY));
    sprintf(fuzFilename, "%s/fort.33", STRINGIZE_VALUE_OF(DATA_DIRECTORY));

    FILE* fux = fopen(fuxFilename, "r");
    FILE* fuy = fopen(fuyFilename, "r");
    FILE* fuz = fopen(fuzFilename, "r");
    if ((!fux)) { printf("Initial conditions file not found\n"); exit(-1); }
    for (unsigned int i=0; i<DIM; i++)
      for (unsigned int j=0; j<DIM; j++)
        for (unsigned l=0; l<DIM; l++)
        {
          fscanf(fux, "%lf ", &(h_ux[i*DIM2+j*DIM+l]));
          fscanf(fuy, "%lf ", &(h_uy[i*DIM2+j*DIM+l]));
          fscanf(fuz, "%lf ", &(h_uz[i*DIM2+j*DIM+l]));
        }
    fclose(fux); fclose(fuy); fclose(fuz);

    thrust::fill(h_uxdt.begin(), h_uxdt.end(), 0.0);
    thrust::fill(h_uydt.begin(), h_uydt.end(), 0.0);
    thrust::fill(h_uzdt.begin(), h_uzdt.end(), 0.0);
    thrust::fill(h_uxx_applied.begin(), h_uxx_applied.end(), 0.0);

    simulation->initialize_simulation(h_ux, h_uy, h_uz, h_uxdt, h_uydt, h_uzdt, h_uxx_applied, ac, as, d2, delta, eo, eta, h, tau);

    printf("Initial conditions set\n");
    cur_timestep=0;
}


//---------------------------------------------------------------------------
// CoGLRender::time_step()
//
// Advance the simulation by calling the CoGLSim class, and report timings
//---------------------------------------------------------------------------
void CoGLRender::time_step()
{
    if (cur_timestep == 0) gettimeofday(&begin, 0);
    
    if (cur_timestep < max_timesteps)
    {
      simulation->advance_simulation(); 
      simulation->color_mesh();
      cur_timestep++;
    }

    if (cur_timestep == max_timesteps)
    {
      gettimeofday(&end, 0);
      timersub(&end, &begin, &diff);
      float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
      std::cout << "Time: " << seconds << std::endl;
    }

}


//---------------------------------------------------------------------------
// CoGLRender::time_simulation()
//
// param    int a_iters    Number of iterations to time   
//
// Time the simulation for a given number of iterations, without rendering
//---------------------------------------------------------------------------
void CoGLRender::time_simulation(int a_iters)
{
    #ifdef USE_INTEROP
      printf("You must disable interop in CMake configuration in order to run timing tests without rendering\n");
    #else
      initialize_simulation();
      cur_timestep = 0;
      max_timesteps = a_iters;
      for (unsigned int i=0; i<max_timesteps; i++) time_step();
    #endif
}



